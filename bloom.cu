#include "hip/hip_runtime.h"
/* 
 * Bloom filter with CUDA.
 *
 * (c) 2019 Josh Kang and Andrew Thai
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "bloom.h"

/*  Change these to test different versions of the CUDA implementation  */

#define SHARED_MAP      // Use for shared copy of filter in Map() kernel
#define SHARED_TEST     // Use for shared copy of miss counter in Test() kernel
#define BLOCK_SIZE 128.0




/* 
 * Uses Horner's rule to get a hash value for a given String.
 */
__device__ unsigned long cuda_hashstring(char *word)
{
  unsigned char *str = (unsigned char *)word;
  unsigned long hash = HASH_NUM;

  while (*str)
  {
    hash = ((hash << 5) + hash) + *(str++);
  }

  return hash;
}

/*
 * Hash string to multiple indices in the Bloom filter.
 */
__device__ void cuda_hash(long *hashes, char *word)
{
  unsigned long x = cuda_hashstring(word);
  unsigned long y = x >> 4;

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    x = (x + y) % M_NUM_BITS;
    y = (y + i) % M_NUM_BITS;
    hashes[i] = x;
  }
}

/*
 * Tests if word is in Bloom filter
 * Unlike the serial version, returns 1 for a miss, 0 for hit
 */
__device__ int cuda_testBloom(unsigned char *filter, char *word)
{
  long hashes[K_NUM_HASH];
  cuda_hash(hashes, word);

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    // miss
    if (!filter[hashes[i]])
    {
      return 1; // +1 for a miss
    }
  }

  return 0;
}

/*
 * Counts number of misses with shared memory.
 */
__global__ void s_cuda_countMisses(unsigned char *filter, String *words, int *count, int num_words)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ int misses;

  // only need one thread to initialize value
  if (threadIdx.x == 0) {
    misses = 0;
  }

  __syncthreads();
  
  if (index < num_words)
  {
    int miss = cuda_testBloom(filter, words[index].word);
    atomicAdd(&misses, miss);
  }

  __syncthreads();
  
  if (threadIdx.x == 0) {
    atomicAdd(count, misses);
  }
}


/*
 * Counts number of misses.
 */
__global__ void cuda_countMisses(unsigned char *filter, String *words, int *count, int num_words)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    int miss = cuda_testBloom(filter, words[index].word);
    atomicAdd(count, miss);
  }
}

/*
 * Set bits in Bloom filter based on hash values.
 */
__device__ void cuda_mapToBloom(unsigned char *filter, char *word)
{
  long hashes[K_NUM_HASH];
  cuda_hash(hashes, word);

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    filter[hashes[i]] = 1;
  }
}

/*
 * Shared version of mapFromArray().
 * Each block has a copy of the Bloom filter.
 * Each thread in a block copies the results to the bloom_filter at the end.
 */
__global__ void s_cuda_mapFromArray(unsigned char *bf_array, String *words, int num_words)
{

  // initialize block's version of Bloom filter
  __shared__ unsigned char s_filter[M_NUM_BITS];

  for (int i = threadIdx.x; i < M_NUM_BITS; i += BLOCK_SIZE) {
    s_filter[i] = 0;
  }
  
  __syncthreads();
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    cuda_mapToBloom(s_filter, words[index].word);
  }

  __syncthreads();

  // copy results into the bloom filter array
  //int chunk = ceil(M_NUM_BITS/BLOCK_SIZE);
  for (int i = threadIdx.x; i < M_NUM_BITS; i += BLOCK_SIZE) {

    // No Atomic functions for unsigned char
    // Use branching to avoid race conditions by only setting when bit is set
    if (s_filter[i]) {
      bf_array[i] = s_filter[i];
    }
  }
  
}

/*
 * Maps elements from the given array to the Bloom filter.
 */
__global__ void cuda_mapFromArray(unsigned char *bf_array, String *words, int num_words)
{

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    cuda_mapToBloom(bf_array, words[index].word);
  }


}


int main(int argc, char **argv)
{

  if (argc != 3)
  {
    printf("Usage: ./bloom WordsToMap WordsTotest\n");
    exit(1);
  }

  // time measurement
  float map_time, test_time = 0;
  hipEvent_t start_map, stop_map, start_test, stop_test;
  checkCudaErrors(hipEventCreate(&start_map));
  checkCudaErrors(hipEventCreate(&stop_map));
  checkCudaErrors(hipEventCreate(&start_test));
  checkCudaErrors(hipEventCreate(&stop_test));

  // open files
  FILE *map_fp = fopen(argv[1], "r");
  if (map_fp == NULL)
  {
    printf("Failed to open file1. \n");
    exit(1);
  }

  FILE *test_fp = fopen(argv[2], "r");
  if (test_fp == NULL)
  {
    printf("Failed to open file2. \n");
    exit(1);
  }

  // host data
  String *h_string_array = (String *)malloc(INIT_WORDS * sizeof(String));
  String *h_test_array = (String *)malloc(INIT_WORDS * sizeof(String));
  int h_misses[1];
  h_misses[0] = 0;

  // read in files
  int num_words_mapped = fileToArray(map_fp, &h_string_array);
  int num_words_test = fileToArray(test_fp, &h_test_array);
  
  // device data
  String *d_string_array;
  String *d_test_array;
  int *d_misses;

  // initialize Bloom filter host and device arrays
  unsigned char *d_bf_array;
  unsigned char *h_bf_array = (unsigned char *)calloc(M_NUM_BITS, sizeof(unsigned char));
  checkCudaErrors(hipMalloc((void **)&d_bf_array, M_NUM_BITS * sizeof(unsigned char)));
  checkCudaErrors(hipMemcpy(d_bf_array, h_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyHostToDevice));

  // allocate device arrays for map kernel
  checkCudaErrors(hipMalloc((void **)&d_string_array, num_words_mapped * sizeof(String)));
  checkCudaErrors(hipMemcpy(d_string_array, h_string_array, num_words_mapped * sizeof(String), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **)&d_test_array, num_words_test * sizeof(String)));
  checkCudaErrors(hipMemcpy(d_test_array, h_test_array, num_words_test * sizeof(String), hipMemcpyHostToDevice));


  // map words to Bloom filter
  checkCudaErrors(hipEventRecord(start_map));
#ifdef SHARED_MAP
  s_cuda_mapFromArray<<<ceil(num_words_mapped / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
                                                                      (String *)d_string_array,
                                                                      num_words_mapped);
#else
  cuda_mapFromArray<<<ceil(num_words_mapped / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
									   (String *)d_string_array,
									   num_words_mapped);
#endif
  checkCudaErrors(hipEventRecord(stop_map));

  // get running time of map
  checkCudaErrors(hipEventSynchronize(stop_map));
  checkCudaErrors(hipEventElapsedTime(&map_time, start_map, stop_map));

  // allocate device data for test kernel
  checkCudaErrors(hipMalloc((void **)&d_misses, sizeof(int)));
  checkCudaErrors(hipMemcpy(d_misses, h_misses, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(h_bf_array, d_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyDeviceToHost));

  // check if words are in Bloom filter
  checkCudaErrors(hipEventRecord(start_test));
#ifdef SHARED_TEST
  s_cuda_countMisses<<<ceil(num_words_test / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
									(String *)d_test_array,
									(int *)d_misses, num_words_test);
#else
  cuda_countMisses<<<ceil(num_words_test / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
									(String *)d_test_array,
									(int *)d_misses, num_words_test);
#endif
  checkCudaErrors(hipEventRecord(stop_test));

  // get running time of test
  checkCudaErrors(hipEventSynchronize(stop_test));
  checkCudaErrors(hipEventElapsedTime(&test_time, start_test, stop_test));

  // get resulting Bloom filter
  //checkCudaErrors(hipMemcpy(h_bf_array, d_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_misses, d_misses, sizeof(int), hipMemcpyDeviceToHost));

  // print run time info
  printInfo(num_words_mapped, num_words_test, map_time, test_time, *h_misses);
  
  // cleanup
  hipEventDestroy(start_map);
  hipEventDestroy(stop_map);
  hipEventDestroy(start_test);
  hipEventDestroy(stop_test);
  hipFree(d_bf_array);
  hipFree(d_string_array);
  hipFree(d_test_array);
  hipFree(d_misses);
  free(h_bf_array);
  free(h_string_array);
  free(h_test_array);

  return 0;
}
