#include "hip/hip_runtime.h"
/*
 * Bloom filter with CUDA.
 *
 * (c) 2019 Josh Kang and Andrew Thai
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "bloom.h"

__device__ unsigned long hashstring(char *word)
{
  unsigned char *str = (unsigned char *)word;
  unsigned long hash = HASH_NUM;

  while (*str)
    {
      hash = ((hash << 5) + hash) + *(str++);
    }

  return hash;
}


__device__ void hash(long *hashes, char *word)
{
  unsigned long x = hashstring(word);
  unsigned long y = x >> 4;

  for (int i = 0; i < K_NUM_HASH; i++)
    {
      x = (x + y) % M_NUM_BITS;
      y = (y + i) % M_NUM_BITS;
      hashes[i] = x;
    }
}

__device__ void mapToBloom(unsigned char *filter, char *word)
{
  long hashes[K_NUM_HASH];
  hash(hashes, word);

  for (int i = 0; i < K_NUM_HASH; i++)
    {
      filter[hashes[i]] = 1;
    }
}

__global__ void addToBloom(unsigned char *bf_array, String *words) 
{
  int index =  blockIdx.x * blockDim.x + threadIdx.x;
  mapToBloom(bf_array, words[index].word);
}

int main(int argc, char** argv) 
{
    
    if (argc != 3) 
    {
        printf("Usage: ./bloom WordsToAdd WordsToCheck\n");
        exit(1);
    }

    // host arrays
    unsigned char *h_bf_array = (unsigned char*)calloc(M_NUM_BITS, sizeof(unsigned char));
    String *h_string_array = (String*)malloc(MAX_WORDS * sizeof(String));
    for (int i = 0; i < MAX_WORDS; i++)
    {
      strcpy(h_string_array[i].word, "");
    }
    
    // device arrays
    unsigned char *d_bf_array;
    String *d_string_array;

    // open files
    FILE *add_fp = fopen(argv[1], "r");
    if (add_fp == NULL)
    {
        printf("Failed to open file1. \n");
        exit(1);
    }

    FILE *check_fp = fopen(argv[2], "r");
    if (check_fp == NULL)
    {
        printf("Failed to open file2. \n");
        exit(1);
    }    
    
    // read in file1
    fileToArray(add_fp, h_string_array);
    
    // allocate device arrays
    checkCudaErrors(hipMalloc((void **) &d_string_array, MAX_WORDS*sizeof(String)));
    checkCudaErrors(hipMemcpy(d_string_array, h_string_array, MAX_WORDS*sizeof(String), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &d_bf_array, M_NUM_BITS*sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(d_bf_array, h_bf_array, M_NUM_BITS*sizeof(unsigned char), hipMemcpyHostToDevice));

    // set dimensions of blocks and grid
    //dim3 dimGrid(ceil(MAX_WORDS/32), 1, 1);
    //dim3 dimBlock(32, 1, 1);
    
    addToBloom<<<16, 32>>>((unsigned char*)d_bf_array, (String*)d_string_array);

    checkCudaErrors(hipMemcpy(h_bf_array, d_bf_array, M_NUM_BITS*sizeof(unsigned char), hipMemcpyDeviceToHost));
    
    for (int i = 0; i < M_NUM_BITS; i++) {
      printf("%d\n", h_bf_array[i]);
    }
    
    hipFree(d_bf_array);
    hipFree(d_string_array);

    free(h_bf_array);
    free(h_string_array);

    return 0;
}
