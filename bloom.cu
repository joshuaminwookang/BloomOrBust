#include "hip/hip_runtime.h"
/* 
 * Bloom filter with CUDA.
 *
 * (c) 2019 Josh Kang and Andrew Thai
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "bloom.h"

#define SHARED
#define BLOCK_SIZE 64.0

/* 
 * Hash String in CUDA.
 */
__device__ unsigned long cuda_hashstring(char *word)
{
  unsigned char *str = (unsigned char *)word;
  unsigned long hash = HASH_NUM;

  while (*str)
  {
    hash = ((hash << 5) + hash) + *(str++);
  }

  return hash;
}

/*
 * Hash string to multiple indices in CUDA.
 */
__device__ void cuda_hash(long *hashes, char *word)
{
  unsigned long x = cuda_hashstring(word);
  unsigned long y = x >> 4;

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    x = (x + y) % M_NUM_BITS;
    y = (y + i) % M_NUM_BITS;
    hashes[i] = x;
  }
}

/*
 * Tests if word is in Bloom filter
 */
__device__ int cuda_testBloom(unsigned char *filter, char *word)
{
  long hashes[K_NUM_HASH];
  cuda_hash(hashes, word);

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    // miss
    if (!filter[hashes[i]])
    {
      return 1; // +1 for a miss
    }
  }

  return 0;
}

/*
 * Counts number of misses.
 */
__global__ void cuda_countMisses(unsigned char *filter, String *words, int *count, int num_words)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    int miss = cuda_testBloom(filter, words[index].word);
    atomicAdd(count, miss);
  }
}

/*
 * Set bits in Bloom filter based on hash values.
 */
__device__ void cuda_mapToBloom(unsigned char *filter, char *word)
{
  long hashes[K_NUM_HASH];
  cuda_hash(hashes, word);

  for (int i = 0; i < K_NUM_HASH; i++)
  {
    filter[hashes[i]] = 1;
  }
}

/*
 * Shared version of mapFromArray().
 * Each block has a copy of the Bloom filter.
 * Each thread in a block copies the results to the bloom_filter at the end.
 */
__global__ void s_cuda_mapFromArray(unsigned char *bf_array, String *words, int num_words)
{

  // initialize block's version of Bloom filter
  __shared__ unsigned char s_filter[M_NUM_BITS];
  memset(s_filter, 0, M_NUM_BITS*sizeof(unsigned char));

  __syncthreads();
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    cuda_mapToBloom(s_filter, words[index].word);
  }

  __syncthreads();

  // copy results into the bloom filter array
  int chunk = ceil(M_NUM_BITS/BLOCK_SIZE);
  for (int i = chunk * threadIdx.x; i < chunk * (threadIdx.x+1) && i < M_NUM_BITS; i++) {

    // avoid race conditions by only setting when bit is set
    if (s_filter[i]) {
      bf_array[i] = s_filter[i];
    }
  }
  
}

/*
 * Maps elements from the given array to the Bloom filter.
 */
__global__ void cuda_mapFromArray(unsigned char *bf_array, String *words, int num_words)
{

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num_words)
  {
    cuda_mapToBloom(bf_array, words[index].word);
  }


}


int main(int argc, char **argv)
{

  if (argc != 3)
  {
    printf("Usage: ./bloom WordsToMap WordsTotest\n");
    exit(1);
  }

  // time measurement
  float map_time, test_time = 0;
  hipEvent_t start_map, stop_map, start_test, stop_test;
  checkCudaErrors(hipEventCreate(&start_map));
  checkCudaErrors(hipEventCreate(&stop_map));
  checkCudaErrors(hipEventCreate(&start_test));
  checkCudaErrors(hipEventCreate(&stop_test));

  // open files
  FILE *map_fp = fopen(argv[1], "r");
  if (map_fp == NULL)
  {
    printf("Failed to open file1. \n");
    exit(1);
  }

  FILE *test_fp = fopen(argv[2], "r");
  if (test_fp == NULL)
  {
    printf("Failed to open file2. \n");
    exit(1);
  }

  // host data
  String *h_string_array = (String *)malloc(INIT_WORDS * sizeof(String));
  String *h_test_array = (String *)malloc(INIT_WORDS * sizeof(String));
  int h_misses[1];
  h_misses[0] = 0;

  // read in files
  int num_words_mapped = fileToArray(map_fp, &h_string_array);
  int num_words_test = fileToArray(test_fp, &h_test_array);
  
  // device data
  String *d_string_array;
  String *d_test_array;
  int *d_misses;

  // initialize Bloom filter host and device arrays
  unsigned char *d_bf_array;
  unsigned char *h_bf_array = (unsigned char *)calloc(M_NUM_BITS, sizeof(unsigned char));
  checkCudaErrors(hipMalloc((void **)&d_bf_array, M_NUM_BITS * sizeof(unsigned char)));
  checkCudaErrors(hipMemcpy(d_bf_array, h_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyHostToDevice));

  // allocate device arrays for map kernel
  checkCudaErrors(hipMalloc((void **)&d_string_array, num_words_mapped * sizeof(String)));
  checkCudaErrors(hipMemcpy(d_string_array, h_string_array, num_words_mapped * sizeof(String), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **)&d_test_array, num_words_test * sizeof(String)));
  checkCudaErrors(hipMemcpy(d_test_array, h_test_array, num_words_test * sizeof(String), hipMemcpyHostToDevice));


  // map words to Bloom filter
  checkCudaErrors(hipEventRecord(start_map));
#ifdef SHARED  
  s_cuda_mapFromArray<<<ceil(num_words_mapped / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
                                                                      (String *)d_string_array,
                                                                      num_words_mapped);
#else
  cuda_mapFromArray<<<ceil(num_words_mapped / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
									   (String *)d_string_array,
									   num_words_mapped);
#endif
  checkCudaErrors(hipEventRecord(stop_map));

  // get running time of map
  checkCudaErrors(hipEventSynchronize(stop_map));
  checkCudaErrors(hipEventElapsedTime(&map_time, start_map, stop_map));

  // allocate device data for test kernel
  checkCudaErrors(hipMalloc((void **)&d_misses, sizeof(int)));
  checkCudaErrors(hipMemcpy(d_misses, h_misses, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(h_bf_array, d_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyDeviceToHost));

  // check if words are in Bloom filter
  checkCudaErrors(hipEventRecord(start_test));
  cuda_countMisses<<<ceil(num_words_test / BLOCK_SIZE), BLOCK_SIZE>>>((unsigned char *)d_bf_array,
                                                                       (String *)d_test_array,
                                                                       (int *)d_misses, num_words_test);
  checkCudaErrors(hipEventRecord(stop_test));

  // get running time of test
  checkCudaErrors(hipEventSynchronize(stop_test));
  checkCudaErrors(hipEventElapsedTime(&test_time, start_test, stop_test));

  // get resulting Bloom filter
  //checkCudaErrors(hipMemcpy(h_bf_array, d_bf_array, M_NUM_BITS * sizeof(unsigned char), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_misses, d_misses, sizeof(int), hipMemcpyDeviceToHost));

  // print run time info
  printInfo(num_words_mapped, num_words_test, map_time, test_time, *h_misses);
  
  // cleanup
  hipEventDestroy(start_map);
  hipEventDestroy(stop_map);
  hipEventDestroy(start_test);
  hipEventDestroy(stop_test);
  hipFree(d_bf_array);
  hipFree(d_string_array);
  hipFree(d_test_array);
  hipFree(d_misses);
  free(h_bf_array);
  free(h_string_array);
  free(h_test_array);

  return 0;
}
